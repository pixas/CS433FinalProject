#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>

#include "mma.h"

using namespace nvcuda;
using std::cout;
using std::endl;
using std::vector;

__global__ void wmma_kernel(half* a, half* b, float* c) {
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

  wmma::fill_fragment(c_frag, 0.0f);
  wmma::load_matrix_sync(a_frag, a, 16);
  wmma::load_matrix_sync(b_frag, b, 16);
  wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
  wmma::store_matrix_sync(c, c_frag, 16, wmma::mem_row_major);
}

void wmma(half* a, half* b, float* c) {}

int main() {
  vector<half> a(16 * 16), b(16 * 16);
  vector<float> c(16 * 16);
  for (int i = 0; i < 16; i++) {
    for (int j = 0; j < 16; j++) {
      if (j >= i) {
        a[i * 16 + j] = __float2half(1);
        b[i + j * 16] = __float2half(1);
      } else {
        a[i * 16 + j] = __float2half(0);
        b[i + j * 16] = __float2half(0);
      }
    }
  }
  int size = 16 * 16;
  half *d_a, *d_b;
  float* d_c;
  hipMalloc((void**)(&d_a), sizeof(half) * size);
  hipMalloc((void**)(&d_b), sizeof(half) * size);
  hipMalloc((void**)(&d_c), sizeof(float) * size);

  hipMemcpy(d_a, a.data(), sizeof(half) * size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), sizeof(half) * size, hipMemcpyHostToDevice);

  dim3 blockDim, gridDim;
  blockDim.x = 4 * 32;
  blockDim.y = 4;
  gridDim.x = 1;
  gridDim.y = 1;
  wmma_kernel<<<gridDim, blockDim>>>(d_a, d_b, d_c);
  hipMemcpy(c.data(), d_c, sizeof(float) * size, hipMemcpyDeviceToHost);
  for (int i = 0; i < 16; i++) {
    for (int j = 0; j < 16; j++) {
      cout << __half2float(a[i * 16 + j]) << ' ';
    }
    cout << endl;
  }
  cout << endl;
  for (int i = 0; i < 16; i++) {
    for (int j = 0; j < 16; j++) {
      cout << c[i * 16 + j] << ' ';
    }
    cout << endl;
  }
}
